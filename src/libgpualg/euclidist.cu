#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "error.hpp"

#include "libgpualg/euclidist.cuh"
#include "libgpualg/mean.cuh"
#include "error.cuh"

__global__ void squared_norm_2_kernel(double *d_A, double *d_B, double *d_res, int pitch, int width, int height, int reducepitch)
{
    int threadid = threadIdx.x; // thread id in the block
    int lineid = blockIdx.y; // rows
    int dataid = blockIdx.x * blockDim.x + threadIdx.x; // cols
    extern __shared__ double s_data[];

    if (dataid >= width || lineid >= height)
    {
        s_data[threadid] = 0;
        return;
    }

    double *d_Aline = (double*)((char*)d_A + lineid * pitch);
    double *d_Bline = (double*)((char*)d_B + lineid * pitch);
    double tmp = d_Aline[dataid] - d_Bline[dataid];
    s_data[threadid] = tmp * tmp;
    __syncthreads();

    for(int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        assert(is_power_of_2(stride));
        if(threadid < stride)
            s_data[threadid] += s_data[threadid + stride];
        __syncthreads();
    }

    double *d_resline = (double*)((char*)d_res + lineid * reducepitch);
    if(threadid == 0)
        d_resline[blockIdx.x] = s_data[0];
}

__host__ double cuda_squared_norm_2(double *d_A, double *d_B, size_t width, size_t height, size_t pitch, int threads)
{
    size_t reducepitch;
    while (!is_power_of_2(threads))
        threads++;
    int nbblocksPerLine = std::ceil((float)width / threads);
    dim3 blocks(nbblocksPerLine, height);

    double *d_res;
    hipMallocPitch(&d_res, &reducepitch, nbblocksPerLine * sizeof(double), height);
    cudaCheckError();
    hipMemset2D(d_res, reducepitch, 0, nbblocksPerLine * sizeof(double), height);
    cudaCheckError();

    // LAUNCHING KERNEL
    std::cerr << "reducepitch: " << reducepitch << "pitch: " << pitch << std::endl;
    std::cerr << "nbthreads: " << threads << " nbblocksPerLine: " << blocks.x << " nbLines: " << blocks.y << std::endl;
    squared_norm_2_kernel<<<blocks, threads, threads * sizeof(double)>>>(d_A, d_B, d_res, pitch, width, height, reducepitch);
    hipDeviceSynchronize();
    cudaCheckError();

    double *h_res = (double*)malloc(height * reducepitch);
    runtime_assert(h_res != nullptr, "Alloc error !");

    // COPY TO HOST
    hipMemcpy(h_res, d_res, height * reducepitch, hipMemcpyDeviceToHost);
    cudaCheckError();

    // FREEING DEVICE MEMORY
    hipFree(d_res);
    cudaCheckError();

    double norm = 0;
    for (size_t i = 0; i < height; ++i)
    {
        double *h_resline = (double*)((char*)h_res + i * reducepitch);
        for (size_t j = 0; j < nbblocksPerLine; ++j)
        {
            norm += h_resline[j];
        }
    }

    return norm;
}
