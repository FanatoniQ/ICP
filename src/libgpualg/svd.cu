#include "libgpualg/svd.cuh"
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <tuple>
#include <cstdio>
#include <cassert>
#include <cstdlib>

void printMatrix(int m, int n, const float* A, int lda, const char* name)
{
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            float Areg = A[row + col * lda];
            printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg);
        }
    }
}

void svd_gpu(float* d_A, size_t r_A, size_t c_A, float *d_U, float *d_S, float *d_VT)
{
    // Error checking variables
    hipsolverHandle_t cusolverH = NULL;

    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    hipError_t cudaStat5 = hipSuccess;
    hipError_t cudaStat6 = hipSuccess;

    // Dimensions
    const int m = r_A;
    const int n = c_A;
    const int lda = m;
    /*
    // Return arrays
    float* U = (float*)malloc(lda * m * sizeof(float));
    if (U == nullptr)
        throw std::bad_alloc();
    float* VT = (float*)malloc(lda * n * sizeof(float));
    if (VT == nullptr)
        throw std::bad_alloc();
    float* S = (float*)malloc(n * sizeof(float));
    if (S == nullptr)
        throw std::bad_alloc();
    //float U[lda * m]; // m-by-m unitary matrix 
    //float VT[lda * n];  // n-by-n unitary matrix
    //float S[n]; // singular value
    */

    //float* d_A = NULL;
    //float* d_S = NULL;
    //float* d_U = NULL;
    //float* d_VT = NULL;
    int* devInfo = NULL;
    float* d_work = NULL;
    float* d_rwork = NULL;
    //float* d_W = NULL;  // W = S*VT

    int lwork = 0;
    int info_gpu = 0;
    /*
    printf("A = (matlab base-1)\n");
    printMatrix(m, n, A, lda, "A");
    printf("=====\n");
    */
    // step 1: create cusolverDn/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    // step 2: copy A and B to device
    //cudaStat1 = hipMalloc((void**)&d_A, sizeof(float) * lda * n);
    //cudaStat2 = hipMalloc((void**)&d_S, sizeof(float) * n);
    //cudaStat3 = hipMalloc((void**)&d_U, sizeof(float) * lda * m);
    //cudaStat4 = hipMalloc((void**)&d_VT, sizeof(float) * lda * n);
    cudaStat5 = hipMalloc((void**)&devInfo, sizeof(int));

    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);
    assert(hipSuccess == cudaStat6);

    //cudaStat1 = hipMemcpy(d_A, A, sizeof(float) * lda * n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

    // step 3: query working space of SVD
    cusolver_status = hipsolverDnDgesvd_bufferSize(
        cusolverH,
        m,
        n,
        &lwork);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(float) * lwork);
    assert(hipSuccess == cudaStat1);

    // step 4: compute SVD
    signed char jobu = 'A'; // all m columns of U
    signed char jobvt = 'A'; // all n columns of VT
    cusolver_status = hipsolverDnDgesvd(
        cusolverH,
        jobu,
        jobvt,
        m,
        n,
        d_A,
        lda,
        d_S,
        d_U,
        lda,  // ldu
        d_VT,
        lda, // ldvt,
        d_work,
        lwork,
        d_rwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);


    //cudaStat1 = hipMemcpy(U, d_U, sizeof(float) * lda * m, hipMemcpyDeviceToHost);
    //cudaStat2 = hipMemcpy(VT, d_VT, sizeof(float) * lda * n, hipMemcpyDeviceToHost);
    //cudaStat3 = hipMemcpy(S, d_S, sizeof(float) * n, hipMemcpyDeviceToHost);
    cudaStat4 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    printf("after gesvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
    printf("=====\n");

    /*
    printf("S = (matlab base-1)\n");
    printMatrix(n, 1, S, lda, "S");
    printf("=====\n");

    printf("U = (matlab base-1)\n");
    printMatrix(m, m, U, lda, "U");
    printf("=====\n");

    printf("VT = (matlab base-1)\n");
    printMatrix(n, n, VT, lda, "VT");
    printf("=====\n");
    */
    // free resources
    //if (d_A) hipFree(d_A);
    //if (d_S) hipFree(d_S);
    //if (d_U) hipFree(d_U);
    //if (d_VT) hipFree(d_VT);
    if (devInfo) hipFree(devInfo);
    if (d_work) hipFree(d_work);
    if (d_rwork) hipFree(d_rwork);
    //if (d_W) hipFree(d_W);

    if (cusolverH) hipsolverDnDestroy(cusolverH);

    //hipDeviceReset();
    //return { U, S, VT };
}
