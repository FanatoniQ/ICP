#include "libgpualg/svd.cuh"
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <tuple>
#include <cstdio>
#include <cassert>
#include <cstdlib>

void printMatrix(int m, int n, const double* A, int lda, const char* name)
{
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            double Areg = A[row + col * lda];
            printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg);
        }
    }
}

std::tuple<double*, double*, double*> svd(double* A, size_t r_A, size_t c_A)
{
    // Error checking variables
    hipsolverHandle_t cusolverH = NULL;

    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    hipError_t cudaStat5 = hipSuccess;
    hipError_t cudaStat6 = hipSuccess;

    // Dimensions
    const int m = r_A;
    const int n = c_A;
    const int lda = m;

    // Return arrays
    double* U = (double*)malloc(lda * m * sizeof(double));
    if (U == nullptr)
        throw std::bad_alloc();
    double* VT = (double*)malloc(lda * n * sizeof(double));
    if (VT == nullptr)
        throw std::bad_alloc();
    double* S = (double*)malloc(n * sizeof(double));
    if (S == nullptr)
        throw std::bad_alloc();
    //double U[lda * m]; // m-by-m unitary matrix 
    //double VT[lda * n];  // n-by-n unitary matrix
    //double S[n]; // singular value
    
    double* d_A = NULL;
    double* d_S = NULL;
    double* d_U = NULL;
    double* d_VT = NULL;
    int* devInfo = NULL;
    double* d_work = NULL;
    double* d_rwork = NULL;
    //double* d_W = NULL;  // W = S*VT

    int lwork = 0;
    int info_gpu = 0;
    
    printf("A = (matlab base-1)\n");
    printMatrix(m, n, A, lda, "A");
    printf("=====\n");

    // step 1: create cusolverDn/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    // step 2: copy A and B to device
    cudaStat1 = hipMalloc((void**)&d_A, sizeof(double) * lda * n);
    cudaStat2 = hipMalloc((void**)&d_S, sizeof(double) * n);
    cudaStat3 = hipMalloc((void**)&d_U, sizeof(double) * lda * m);
    cudaStat4 = hipMalloc((void**)&d_VT, sizeof(double) * lda * n);
    cudaStat5 = hipMalloc((void**)&devInfo, sizeof(int));
    //cudaStat6 = hipMalloc((void**)&d_W, sizeof(double) * lda * n);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);
    assert(hipSuccess == cudaStat6);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

    // step 3: query working space of SVD
    cusolver_status = hipsolverDnDgesvd_bufferSize(
        cusolverH,
        m,
        n,
        &lwork);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double) * lwork);
    assert(hipSuccess == cudaStat1);

    // step 4: compute SVD
    signed char jobu = 'A'; // all m columns of U
    signed char jobvt = 'A'; // all n columns of VT
    cusolver_status = hipsolverDnDgesvd(
        cusolverH,
        jobu,
        jobvt,
        m,
        n,
        d_A,
        lda,
        d_S,
        d_U,
        lda,  // ldu
        d_VT,
        lda, // ldvt,
        d_work,
        lwork,
        d_rwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);


    cudaStat1 = hipMemcpy(U, d_U, sizeof(double) * lda * m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(VT, d_VT, sizeof(double) * lda * n, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(S, d_S, sizeof(double) * n, hipMemcpyDeviceToHost);
    cudaStat4 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    printf("after gesvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
    printf("=====\n");

    printf("S = (matlab base-1)\n");
    printMatrix(n, 1, S, lda, "S");
    printf("=====\n");

    printf("U = (matlab base-1)\n");
    printMatrix(m, m, U, lda, "U");
    printf("=====\n");

    printf("VT = (matlab base-1)\n");
    printMatrix(n, n, VT, lda, "VT");
    printf("=====\n");

    // free resources
    if (d_A) hipFree(d_A);
    if (d_S) hipFree(d_S);
    if (d_U) hipFree(d_U);
    if (d_VT) hipFree(d_VT);
    if (devInfo) hipFree(devInfo);
    if (d_work) hipFree(d_work);
    if (d_rwork) hipFree(d_rwork);
    //if (d_W) hipFree(d_W);

    if (cusolverH) hipsolverDnDestroy(cusolverH);

    hipDeviceReset();
    return { U, S, VT };
}
