#include "hip/hip_runtime.h"
#include <vector>
#include <limits>
#include <tuple>
#include <iostream>
#include <cmath>

#include "libalg/basic_operations.hpp"
#include "libalg/alg.hpp"
#include "libalg/CPUMatrix.hpp"
//#include "cpu/icp.hpp"
#include "libalg/CPUView.hpp"
#include "error.hpp"
#include "cpu/tuple.hpp"

#include "gpu/icp.cuh"

#define Tile_size 2

/* --------- CPU Version Calling GPU Kernel ------------ */
__host__ std::vector<std::tuple<size_t, int>> get_correspondence_indices(double *P, double *Q,
                                                                size_t P_r, size_t P_c, size_t Q_r, size_t Q_c)
{
    std::vector<std::tuple<size_t, int>> correspondances = {};
    for (size_t i = 0; i < P_r; i++)
    {
        double *p_point = P + i * P_c;
        double min_dist = std::numeric_limits<double>::max();
        int chosen_idx = -1;
        for (size_t j = 0; j < Q_r; j++)
        {
            double *q_point = Q + j * Q_c;
            double dist = std::sqrt(element_wise_reduce(p_point, q_point, 1, P_c, 1, Q_c,
                                    squared_norm_2, add, add)); //norm 2 between 2 vectors
            if (dist < min_dist)
            {
                min_dist = dist;
                chosen_idx = j;
            }
        }
        correspondances.push_back(std::make_tuple(i, chosen_idx));
    }
    return correspondances;
}


// Intermediation function to be replaced with element_wise_op
__host__ void increment_cov(double *P, double *Q)
{
    for (int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 3; j++)
        {
            P[i*3 + j] = P[i*3 + j] + Q[i*3 + j];
        }
    }
}

__host__ double* calling_transpose_kernel(double *A, size_t row, size_t column)
{
        // Calling transpose kernel
        size_t size = sizeof(double) * row * column;

        // Allocations
        double *d_source_transpose, *d_dest_transpose;
        hipMalloc((void **)&d_source_transpose, size);
        hipMalloc((void **)&d_dest_transpose, size);
        double *transposed_Q = (double *)calloc(size, sizeof(double));

        // Copy mem and exec 
        hipMemcpy(d_source_transpose, A, size, hipMemcpyHostToDevice);
        gpuTranspose(d_source_transpose, d_dest_transpose, row, column);
        hipMemcpy(transposed_Q, d_dest_transpose, size, hipMemcpyDeviceToHost);
        
        // Free cuda mem
        hipFree(d_source_transpose);
        hipFree(d_dest_transpose);

        // End of transpose call
        return transposed_Q;
}

__host__ double *compute_cross_variance_cpu_call_gpu(double *P, double *Q, std::vector<std::tuple<size_t, int>> correspondences, size_t P_r, size_t P_c,
                                size_t Q_r, size_t Q_c) //set default function to lambda function??
{
    UNUSED(Q_r);
    double *cov = (double *)calloc(9, sizeof(double));

    for (auto tup : correspondences)
    {
        auto i = std::get<0>(tup);
        auto j = std::get<1>(tup);
        double *q_point = Q + j * Q_c;
        double *p_point = P + i * P_c;

        double *doted_points = nullptr;
        
        double *transposed_Q = calling_transpose_kernel(q_point, 1, Q_c);
        //double *transposed_Q = transpose(q_point, 1, Q_c);

        dot_product(&doted_points, transposed_Q, p_point, Q_c, 1, 1, P_c); //dim of Q_r * P_r
        free (transposed_Q); 
        increment_cov(cov, doted_points); //need to set element_wise_op but too complicated, doesn't work for some reason.
        free(doted_points);
    }
    return cov;
}

/* -------------- Version GPU Kernel -----------*/

// Implementation with double arrays and no vector for full GPU usage
__global__ void get_correspondence_indices_array_gpu(tuple **correspondances, double *P, double *Q, size_t P_r, size_t P_c, size_t Q_r, size_t Q_c)
{
    int push_index = 0;
    for (size_t i = 0; i < P_r; i++)
    {
        double *p_point = P + i * P_c;
        double min_dist = std::numeric_limits<double>::max();
        int chosen_idx = -1;
        for (size_t j = 0; j < Q_r; j++)
        {
            double *q_point = Q + j * Q_c;
            double dist = std::sqrt(*p_point + *q_point);
            //double dist = std::sqrt(element_wise_reduce(p_point, q_point, 1, P_c, 1, Q_c,
            //                        squared_norm_2, add, add)); //norm 2 between 2 vectors
            if (dist < min_dist)
            {
                min_dist = dist;
                chosen_idx = j;
            }
        }
        tuple *new_tup = nullptr;
        hipMalloc(&new_tup, sizeof(tuple));
        //tuple *new_tup = (tuple*)calloc(1, sizeof(tuple));
        new_tup->index = i;
        new_tup->value = chosen_idx;
        correspondances[push_index] = new_tup;
        push_index++;
    }
}

// Array implementation for GPU
void compute_cross_variance_array(double * cov, double *P, double *Q, std::tuple<size_t, int> *correspondences, size_t P_r, size_t P_c,
                                size_t Q_r, size_t Q_c) //set default function to lambda function??
{
    UNUSED(Q_r);
    UNUSED(P_r);

    for (size_t index = 0; index < P_r; index ++)
    {
        auto i = std::get<0>(correspondences[index]);
        auto j = std::get<1>(correspondences[index]);
        double *q_point = Q + j * Q_c;
        double *p_point = P + i * P_c;

        double *transposed_Q = transpose(q_point, 1, Q_c);
        double *doted_points = nullptr;
        dot_product(&doted_points, transposed_Q, p_point, Q_c, 1, 1, P_c); //dim of Q_r * P_r
        free (transposed_Q); 
        increment_cov(cov, doted_points); //need to set element_wise_op but too complicated, doesn't work for some reason.
        free(doted_points);
    }
}

__global__ void naiveGPUTranspose(const double *d_a, double *d_b, const int rows, const int cols) 
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int index_in = i * cols + j;
    int index_out = j * rows + i;

    if (i < rows && j < cols)
        d_b[index_out] = d_a[index_in];
}

void gpuTranspose(double* A, double* B, int numRows, int numColumns) {

    // declare the number of blocks per grid and the number of threads per block
    dim3 dimGrid((numColumns / Tile_size), (numRows / Tile_size), 1);//Number of Blocks required
    dim3 dimBlock(Tile_size, Tile_size, 1);//Number of threads in each block

    //@@ Launch the GPU Kernel here
    naiveGPUTranspose<<<dimGrid, dimBlock>>>(A, B, numRows, numColumns);
}
