#include "gpu/gputimer.cuh"

GPUTimer::~GPUTimer()
{
    hipEventDestroy(GPUTimer::start);
    hipEventDestroy(GPUTimer::stop);
}

void GPUTimer::Start()
{
    hipEventRecord(GPUTimer::start);
}

void GPUTimer::Stop()
{
    hipEventRecord(GPUTimer::stop);
}

float GPUTimer::ElapsedTime()
{
    float elapsedTime;
    hipEventSynchronize(GPUTimer::stop);
    hipEventElapsedTime(&elapsedTime, GPUTimer::start, GPUTimer::stop);
    return elapsedTime;
}
