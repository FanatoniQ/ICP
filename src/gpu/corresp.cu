#include "hip/hip_runtime.h"
#include <limits>
#include <float.h>
#include <assert.h>
#include <stdio.h>

#include "libgpualg/mean.cuh"
#include "gpu/corresp.cuh"
#include "error.cuh"

#define is_power_of_2(x) (x & (x-1)) == 0

__global__ void get_correspondences_kernel(ICPCorresp *d_dist,
    size_t dist_pitch, size_t dist_0, size_t dist_1)
{
    extern __shared__ ICPCorresp s_data[]; // s_data is of size blockDim.x
    unsigned int threadid = threadIdx.x; // thread id in the block
    unsigned int lineid = blockIdx.y; // line
    unsigned int dataid = blockIdx.x * blockDim.x + threadIdx.x; // column
    if (dataid >= dist_1 || lineid >= dist_0) {
        s_data[threadid] = { DBL_MAX,dataid };
        return;
    }
    // each thread copies to shared memory
    ICPCorresp *d_distline = (ICPCorresp *)((char *)d_dist + lineid * dist_pitch);
    s_data[threadid] = d_distline[dataid];
    __syncthreads();
    // each thread will reduce with one other shared data element in the middle right part of s_data
    for (size_t stride = blockDim.x / 2; stride > 0; stride = stride >> 1)
    {
        assert(is_power_of_2(stride)); // if not power of 2 ...
        if (threadid < stride) { // a lot of threads are idle...
            if (s_data[threadid + stride].dist < s_data[threadid].dist)
                s_data[threadid] = s_data[threadid + stride];
        }
        __syncthreads();
    }
    if (threadid == 0)
        d_distline[blockIdx.x] = s_data[0]; // or [0] since gridsize.x should be 1
}

__host__ void get_correspondences(ICPCorresp *d_dist,
    size_t dist_pitch, size_t dist_0, size_t dist_1, bool sync)
{
    dim3 gridsize(1, get_next_power_of_2(dist_0));
    dim3 blocksize(dist_1, 1);
    get_correspondences_kernel<<<gridsize, blocksize, blocksize.x * sizeof(ICPCorresp)>>>(d_dist, dist_pitch, dist_0, dist_1);
    if (sync) {
        hipDeviceSynchronize();
        cudaCheckError();
    }
}
