#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <tuple>
#include <iostream>
#include <limits>
#include <float.h>

// CPU
#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"
#include "error.hpp"


// GPU
#include "libgpualg/mean.cuh"
#include "error.cuh"
#include "gpu/icp.cuh"
#include "gpu/corresp.cuh"
#include "gpu/dist.cuh"
#include "gpu/corresp.cuh"

__host__ double *get_cross_covs_cpu(const CPUMatrix &P, size_t p_0, size_t p_1,
    const CPUMatrix &Q, size_t q_0, size_t q_1,
    ICPCorresp *d_dist, size_t dist_0, size_t dist_1, size_t dist_pitch)
{
    size_t ref_pitch = q_1 * p_1 * sizeof(double);
    double *h_ref = (double*)malloc(p_0 * ref_pitch);

    size_t h_dist_pitch = dist_1 * sizeof(ICPCorresp);
    ICPCorresp *h_dist = (double*)malloc(dist_0 * h_dist_pitch);

    hipMemcpy2D(h_dist, h_dist_pitch, d_dist, dist_pitch, dist_1 * sizeof(ICPCorresp), dist_0, hipMemcpyDeviceToHost);
    cudaCheckError();

    for (size_t i; i < p_0; ++i)
    {
        size_t idq = h_dist[i].id;
        auto cov = Q.getLine(idq).dot(P.getLine(i).transpose());
        assert(ref_pitch == cov.getDim0() * cov.getDim1());
        memcpy(h_ref + i * ref_pitch, cov.getArray(), cov.getDim0() * cov.getDim1() * sizeof(double));
    }

    return h_ref;
}

int main(int argc, char **argv)
{
    std::string f1Header{};
    size_t Qlines, Qcols, Plines, Pcols;
    double *Pt = readCSV(argv[1], f1Header, Plines, Pcols);
    CPUMatrix P = CPUMatrix(Pt, Plines, Pcols);
    double *Qt = readCSV(argv[2], f1Header, Qlines, Qcols);
    CPUMatrix Q = CPUMatrix(Qt, Qlines, Qcols);

    // device P matrix
    size_t p_pitch = Pcols * sizeof(double);
    double *d_P;
    //cudaMallodist_pitch((void **)&d_P, &p_pitch, Pcols * sizeof(double), Plines);
    hipMalloc((void**)&d_P, Plines * p_pitch);
    cudaCheckError();
    hipMemcpy2D(d_P, p_pitch, Pt, Pcols * sizeof(double), Pcols * sizeof(double), Plines, hipMemcpyHostToDevice);
    cudaCheckError();

    // device Q matrix
    size_t q_pitch = Qcols * sizeof(double);
    double *d_Q;
    //cudaMallodist_pitch((void **)&d_Q, &q_pitch, Qcols * sizeof(double), Qlines);
    hipMalloc((void**)&d_Q, Qlines * q_pitch);
    cudaCheckError();
    hipMemcpy2D(d_Q, q_pitch, Qt, Qcols * sizeof(double), Qcols * sizeof(double), Qlines, hipMemcpyHostToDevice);
    cudaCheckError();

    // device dist,id distance and corresp matrix
    size_t dist_0 = Plines, dist_1 = Qlines;
    size_t dist_pitch;
    ICPCorresp *d_dist;
    cudaMallodist_pitch((void **)&d_dist, &dist_pitch, dist_1 * sizeof(ICPCorresp), dist_0);
    cudaCheckError();
    //hipMemcpy2D(d_dist, dist_pitch, C, Qlines * sizeof(ICPCorresp), Qlines * sizeof(ICPCorresp), Plines, hipMemcpyHostToDevice);
    //cudaCheckError();

    // device cross-covs flattened
    size_t Rlines = Plines, Rcols = Pcols * Qcols;
    size_t r_pitch = Rcols * sizeof(double);
    double *d_R;
    // or 2d...
    hipMalloc((void**)&d_R, Rlines * r_pitch);
    cudaCheckError();

    // DISTS
    get_distances(d_P, d_Q, &d_dist, Plines, Pcols, p_pitch, Qlines, Qcols, q_pitch, Plines, Qlines, &dist_pitch, true);
    std::cerr << "DISTS DONE" << std::endl;

    // CORRESPS
    get_correspondences(d_dist, dist_pitch, dist_0, dist_1, true);
    std::cerr << "CORRESPS DONE" << std::endl;

    /** Testing corresps: **/
    //ICPCorresp *h_res = (ICPCorresp *)malloc(Plines * Qlines * sizeof(ICPCorresp));
    //hipMemcpy2D(h_res, Qlines * sizeof(ICPCorresp), d_dist, dist_pitch, 1 * sizeof(ICPCorresp), Plines, hipMemcpyDeviceToHost);
    //cudaCheckError();

    // CROSS-COVS
    get_cross_cov(d_P, d_Q, &d_R, d_dist,
        Plines, Pcols, p_pitch,
        Qlines, Qcols, q_pitch,
        Rlines, Rcols, &r_pitch,
        dist_0, dist_1, dist_pitch, true);
    std::cerr << "CROSS-COVS DONE" << std::endl;

    /** Testing cross-covs: **/
    double *h_ref_cross_covs = get_cross_covs_cpu(P, Plines, Pcols, Q, Qlines, Qcols, d_dist, dist_0, dist_1, dist_pitch);
    double *h_r = (double*)malloc(Rlines * Rcols * sizeof(double));
    hipMemcpy2D(h_r, Rcols * sizeof(double), d_dist, dist_pitch, Rcols * sizeof(double), Rlines, hipMemcpyDeviceToHost);
    cudaCheckError();
    assert(memcmp(h_ref_cross_covs, h_r, Rlines * Rcols * sizeof(double)) == 0);
    
    free(h_r);
    free(h_ref_cross_covs);
    hipFree(d_P);
    cudaCheckError();
    hipFree(d_Q);
    cudaCheckError();
    hipFree(d_R);
    cudaCheckError();
    hipFree(d_dist);
    cudaCheckError();
}