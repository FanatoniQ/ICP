#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <iomanip>

#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"

__global__ void print_kernel()
{
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

// TODO: REMOVE ME since useless
__global__ void print_matrix_kernel(char *d_A, int pitch, int nbvals)
{
    int j;
    int idx = threadIdx.x;
    double *line = (double*)(d_A + idx * pitch);
    printf("Line %d:\n", idx);
    for (j = 0; j < nbvals; ++j) {
        //printf("%6.2f\t", (double)(d_A[idx * pitch + j * sizeof(double)]));
        printf("%6.2f\t", line[j]);
	__syncthreads();
    }
    printf("\n");
}

// computes line sum
__device__ __host__ double get_line_sum(const double *line, int nbvals)
{
    int j;
    double sum = 0;
    for (j = 0; j < nbvals; ++j)
        sum += line[j];
    return sum;
}

// computes line mean
__device__ __host__ double get_line_mean(const double *line, int nbvals)
{
    return get_line_sum(line, nbvals) / nbvals;
}

/**
 ** \brief dumb_sum_kernel a dummy kernel to compute axis=1 sum
 ** each thread will sum one line with coallesced access then write
 ** the sum to global device memory at address d_sumA[idx] with
 ** idx being the threadIdx.x. In order to get the sum for each lines
 ** we need to launch nblines kernels. Each thread will call the
 ** get_line_sum function.
 ** \note kernel can be improved by launching multiple threads per line
 ** and potentially divide and conquer strategy
 **
 ** \param d_A the device pointer
 ** \param d_sumA the device sum pointer used to store sum results
 ** \param pitch the pitch in bytes
 ** \param nbvals the number of values in a line
 **/
__global__ void dumb_sum_kernel(const char *d_A, double *d_sumA, int pitch, int nbvals)
{
    //int j;
    int idx = threadIdx.x;
    //double sum = 0;
    double *line = (double *)(d_A + idx * pitch);
    //for (j = 0; j < nbvals; ++j)
    //    sum += line[j];
    //d_sumA[idx] = sum;
    d_sumA[idx] = get_line_sum(line, nbvals);
}

/**
 ** \brief dumb_mean_kernel a dummy kernel to compute axis=1 mean
 ** This kernel is basically the same as the dumb_sum_kernel, but
 ** each thread calls the get_line_mean function instead of get_line_sum.
 ** In order to get the mean for each lines we need to launch nblines kernels.
 **
 ** \param d_A the device pointer
 ** \param d_meanA the device mean pointer used to store mean results
 ** \param pitch the pitch in bytes
 ** \param nbvals the number of values in a line
 **/
__global__ void dumb_mean_kernel(const char *d_A, double *d_meanA, int pitch, int nbvals)
{
    int idx = threadIdx.x;
    double *line = (double *)(d_A + idx * pitch);
    d_meanA[idx] = get_line_mean(line, nbvals); //get_line_sum(line, nbvals) / nbvals;
}

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        std::cerr << "Usage: ./hello file1" << std::endl;
        exit(1);
    }
    std::cerr << std::setprecision(15);
    std::string h{};
    size_t nblines, nbcols;
    double *h_p = readCSV(argv[1], h, nblines, nbcols);
    double *h_pT = transpose(h_p, nblines, nbcols);
    print_matrix(std::cerr, h_p, nbcols, nblines);

    auto P = CPUMatrix(h_pT, nbcols, nblines);
    std::cerr << "CPU Means: " << std::endl << P.mean(1) << P.mean(0) << std::endl;

    //print_kernel<<<2, 3>>>();

    // device memory
    double *d_pT;
    size_t pitch;
    size_t width = nblines, height = nbcols;
    hipMallocPitch(&d_pT, &pitch, width * sizeof(double), height * sizeof(double)); // TODO: error checking
    hipMemcpy2D(d_pT, pitch, h_pT, width * sizeof(double), width * sizeof(double), height, hipMemcpyHostToDevice); // TODO: error checking
    double *d_mean;
    hipMalloc(&d_mean, height * sizeof(double)); // TODO: error checking

    // kernel launching
    //print_matrix_kernel<<<1, height>>>((char *)d_pT, pitch, width);
    //dumb_sum_kernel<<<1, height>>>((char*)d_pT, d_mean, pitch, width);
    dumb_mean_kernel<<<1, height>>>((char*)d_pT, d_mean, pitch, width);
    hipDeviceSynchronize();

    // copy back to host memory
    double *h_mean;
    h_mean = (double*)malloc(height * sizeof(double));
    if (h_mean == nullptr)
        exit(1);
    hipMemcpy(h_mean, d_mean, height * sizeof(double), hipMemcpyDeviceToHost);

    std::cerr << "GPU Mean : " << std::endl;
    for (size_t i = 0; i < height; ++i)
        std::cerr << h_mean[i] << "\t";

    hipFree(d_mean);
    hipFree(d_pT);
    free(h_mean);
    free(h_p);
    //free(h_pT);
    return EXIT_SUCCESS;
}
