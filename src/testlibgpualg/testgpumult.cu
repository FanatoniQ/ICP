#include "hip/hip_runtime.h"
#include "libgpualg/mult.cuh"
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>

//Normal CPU Matrix Multiplication
void matMultiplyOnHost(double* A, double* B, double* C, int numARows,
    int numAColumns, int numBRows, int numBColumns,
    int numCRows, int numCColumns)
{
    for (int i = 0; i < numARows; i++)
    {
        for (int j = 0; j < numAColumns; j++)
        {
            C[i * numCColumns + j] = 0;
            for (int k = 0; k < numCColumns; k++)
            {
                C[i * numCColumns + j] += A[i * numAColumns + k] * B[k * numBColumns + j];
            }
        }
    }
    return;
}

void print_Mat(int Row, int Col, double* Mat)
{
    for (int i = 0; i < Row * Col; i++)
    {
        printf("%f  ", *(Mat + i));

        if ((i % Col) == 0)
        {
            printf("\n");
        }
    }
}

int main(int argc, char** argv)
{
    // Perform matrix multiplication C = A*B
    int h_A_row = 1;
    int h_A_col = 3;
    int h_B_row = 3;
    int h_B_col = 3;
    int h_C_row = h_A_row;
    int h_C_col = h_B_col;

    // Allocate memory on the host
    double* h_A = (double*)malloc(h_A_row * h_A_col * sizeof(double));
    double* h_B = (double*)malloc(h_B_row * h_B_col * sizeof(double));
    double* h_C = (double*)malloc(h_C_row * h_C_col * sizeof(double));

    for (int i = 0; i < h_A_row; i++) {
        for (int j = 0; j < h_A_col; j++) {
            h_A[i * h_A_row + j] = 2;//sin(i);
        }
    }
    h_A[2] = 9.;
    for (int i = 0; i < h_B_row; i++) {
        for (int j = 0; j < h_B_col; j++) {
            h_B[i * h_B_row + j] = 2;//sin(i);
        }
    }
    h_B[6] = 53.;

    double *d_A;
    double *d_B;
    double* d_C;
    hipMalloc(&d_A, h_A_row * h_A_col * sizeof(double));
    hipMalloc(&d_B, h_B_row * h_B_col * sizeof(double));
    hipMalloc(&d_C, h_C_row * h_C_col * sizeof(double));

    hipMemcpy(d_A, h_A, h_A_row * h_A_col * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, h_B_row * h_B_col * sizeof(double), hipMemcpyHostToDevice);

    matrixMultiplication(d_A, d_B, d_C, h_A_row, h_A_col, h_B_row, h_B_col, h_C_row, h_C_col);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, h_C_row * h_C_col * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    double* cpu_C;
    cpu_C = new double[h_C_row * h_C_col];

    matMultiplyOnHost(h_A, h_B, cpu_C, h_A_row, h_A_col, h_B_row, h_B_col, h_C_row, h_C_col);

    double err = 0;
    // Check the result and make sure it is correct
    for (int i = 0; i < h_C_col * h_C_row; i++) {
        err += cpu_C[i] - h_C[i];
        if (cpu_C[i] != h_C[i])
        {
            printf("Mismatch at Row = %d Col = %d hostComputed[] = %f --device[] %f\n", i / h_C_col, i % h_C_col, cpu_C[i], h_C[i]);
            break;
        }
    }
    hipDeviceSynchronize();
    std::cerr << "Error: " << err << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipDeviceReset();

    return 0;
}