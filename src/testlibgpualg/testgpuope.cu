#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <iomanip>

// CPU
#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"
#include "libalg/broadcasting.hpp"
#include "error.hpp"

// GPU
#include "error.cuh"
#include "libgpualg/ope.cuh"

// TODO: export this in static lib, was linking failing or invalid device pointer

template <typename T> 
__device__
T add2(T a, T b)
{
    return a + b;
}

template <typename T> 
__device__
T subtract2(T a, T b)
{
    return a - b;
}

template <typename T> 
__device__
T mult2(T a, T b)
{
    return a * b;
}

template <typename T> 
__device__
T divide2(T a, T b)
{
    return a / b;
}

// explicit pointer instanciation for use in kernel...
// TODO: export this in static lib, was linking failing or invalid device pointer
// we could use constant memory function table array in static lib for exemple

template <typename T>
__device__ func2_t<T> add2_op = add2<T>;

template <typename T>
__device__ func2_t<T> subtract2_op = subtract2<T>;

template <typename T>
__device__ func2_t<T> mult2_op = mult2<T>;

template <typename T>
__device__ func2_t<T> divide2_op = divide2<T>;

// MAIN

int main(int argc, char **argv)
{
    runtime_assert(argc == 4, "Usage: ./testgpuope file1 meanaxis op");
    
    // retrieving functions (this part is not required if not on __host__ function)
    //func2_t<double> h_add2_op, h_subtract2_op, h_mult2_op, h_divide2_op;
    /**
    hipMemcpyFromSymbol(&h_add2_op, HIP_SYMBOL(add2_op<double>), sizeof(func2_t<double>));
    cudaCheckError();
    hipMemcpyFromSymbol(&h_subtract2_op, HIP_SYMBOL(subtract2_op<double>), sizeof(func2_t<double>));
    cudaCheckError();
    hipMemcpyFromSymbol(&h_mult2_op, HIP_SYMBOL(mult2_op<double>), sizeof(func2_t<double>));
    cudaCheckError();
    hipMemcpyFromSymbol(&h_divide2_op, HIP_SYMBOL(divide2_op<double>), sizeof(func2_t<double>));
    cudaCheckError();
    **/
    // TODO:
    func2_t<double> h_op;

    // reading file, cpu operations
    std::string h{};
    size_t nblines, nbcols;
    double *h_A = readCSV(argv[1], h, nblines, nbcols);
    std::cerr << nblines << nbcols << std::endl;
    auto A = CPUMatrix(h_A, nblines, nbcols);
    std::cerr << A << std::endl;
    int axis = std::stoi(argv[2]); 
    auto cpuMean = A.mean(axis);
    // transpose if axis is 1 
    if (axis == 1)
        cpuMean = cpuMean.transpose();

    // left operand
    double *d_A;
    size_t d_apitch;
    unsigned int a_0 = A.getDim0(), a_1 = A.getDim1(); //size_t width = nbcols, height = nblines;
    hipMallocPitch(&d_A, &d_apitch, a_1 * sizeof(double), a_0 * sizeof(double));
    cudaCheckError();
    hipMemcpy2D(d_A, d_apitch, A.getArray(), a_1 * sizeof(double), a_1 * sizeof(double), a_0, hipMemcpyHostToDevice);
    cudaCheckError();

    // right operand
    double *d_B;
    size_t d_bpitch;
    unsigned int b_0 = cpuMean.getDim0(), b_1 = cpuMean.getDim1();
    hipMallocPitch(&d_B, &d_bpitch, b_1 * sizeof(double), b_0 * sizeof(double));
    cudaCheckError();
    hipMemcpy2D(d_B, d_bpitch, cpuMean.getArray(), b_1 * sizeof(double), b_1 * sizeof(double), b_0, hipMemcpyHostToDevice);
    cudaCheckError();

    // result
    double *d_R = d_A; // in place operation
    size_t d_rpitch = d_apitch;
    size_t r_0, r_1;
    runtime_assert(get_broadcastable_size(a_0, a_1, b_0, b_1, &r_0, &r_1), "Invalid size for broadcasting !");
    runtime_assert(r_0 == a_0 && r_1 == a_1, "Invalid broadcasting for inplace operation !");

    // Launch the kernel
    dim3 blocksize(32,32); // 1024 threads per block TODO: change to test
    int nbblocksx = std::ceil((float)r_1 / blocksize.x);
    int nbblocksy = std::ceil((float)r_0 / blocksize.y);
    dim3 gridsize(nbblocksx, nbblocksy);
    runtime_assert(gridsize.x * gridsize.y * blocksize.x * blocksize.y >= r_0 * r_1, "Not enough threads !");
    //int threads = 4; // TODO: change this
    //int blocks = std::ceil((float)r_0 * r_1 / threads);
    //dim3 blocks(nbblocks, height);
    //broadcast_op_kernel<double><<<gridsize, blocksize>>>(d_A, d_B, d_R, h_subtract2_op,
    std::cerr << d_apitch << std::endl;
    std::cerr << d_bpitch << std::endl;
    std::cerr << b_0 << "," << b_1 << std::endl;
    //broadcast_subtract_kernel<<<gridsize, blocksize>>>(d_A, d_B, d_R,
    //auto R = A - cpuMean;
    
    //runtime_assert(R.getArray() == nullptr, "Not standard empty init CPUMatrix behaviour");
    if (strcmp(argv[3], "-") == 0)
    {
         A -= cpuMean; // testing centered data
	 hipMemcpyFromSymbol(&h_op, HIP_SYMBOL(subtract2_op<double>), sizeof(func2_t<double>));
         cudaCheckError();
/**broadcast_op_kernel<double><<<gridsize, blocksize>>>(d_A, d_B, d_R, h_subtract2_op,
        a_0, a_1, d_apitch / sizeof(double),
        b_0, b_1, d_bpitch / sizeof(double),
        r_0, r_1, d_rpitch / sizeof(double));**/
    }
    else if (strcmp(argv[3], "+") == 0)
    {
         A += cpuMean;
	 hipMemcpyFromSymbol(&h_op, HIP_SYMBOL(add2_op<double>), sizeof(func2_t<double>));
         cudaCheckError();
/**broadcast_op_kernel<double><<<gridsize, blocksize>>>(d_A, d_B, d_R, h_add2_op,
        a_0, a_1, d_apitch / sizeof(double),
        b_0, b_1, d_bpitch / sizeof(double),
        r_0, r_1, d_rpitch / sizeof(double));**/
    }
    else if (strcmp(argv[3], "x") == 0)
    {
         A *= cpuMean;
	 hipMemcpyFromSymbol(&h_op, HIP_SYMBOL(mult2_op<double>), sizeof(func2_t<double>));
         cudaCheckError();
/**broadcast_op_kernel<double><<<gridsize, blocksize>>>(d_A, d_B, d_R, h_mult2_op,
        a_0, a_1, d_apitch / sizeof(double),
        b_0, b_1, d_bpitch / sizeof(double),
        r_0, r_1, d_rpitch / sizeof(double));**/
    }
    else if (strcmp(argv[3], "/") == 0)
    {
         A /= cpuMean;
	 hipMemcpyFromSymbol(&h_op, HIP_SYMBOL(divide2_op<double>), sizeof(func2_t<double>));
         cudaCheckError();
/**broadcast_op_kernel<double><<<gridsize, blocksize>>>(d_A, d_B, d_R, h_divide2_op,
        a_0, a_1, d_apitch / sizeof(double),
        b_0, b_1, d_bpitch / sizeof(double),
        r_0, r_1, d_rpitch / sizeof(double));**/
    }
    else
    {
        std::cerr << "Invalid op" << std::endl;
        return EXIT_FAILURE;
    }

    /**broadcast_subtract_kernel<<<gridsize, blocksize>>>(d_A, d_B, d_R,
        a_0, a_1, d_apitch / sizeof(double),
        b_0, b_1, d_bpitch / sizeof(double),
        r_0, r_1, d_rpitch / sizeof(double));**/
    if (b_0 == 1 && b_1 == 1) {
        broadcast_op_scalar_kernel<double><<<gridsize, blocksize>>>(d_A, d_B, d_R, h_op,
            a_0, a_1, d_apitch / sizeof(double),
            r_0, r_1, d_rpitch / sizeof(double));
    } else if (b_0 == 1) {
        broadcast_op_line_vector_kernel<double><<<gridsize, blocksize, blocksize.x * sizeof(double)>>>(d_A, d_B, d_R, h_op,
            a_0, a_1, d_apitch / sizeof(double),
            b_0, b_1, d_bpitch / sizeof(double),
            r_0, r_1, d_rpitch / sizeof(double));
    } else if (b_1 == 1) {
        broadcast_op_column_vector_kernel<double><<<gridsize, blocksize, blocksize.x * sizeof(double)>>>(d_A, d_B, d_R, h_op,
            a_0, a_1, d_apitch / sizeof(double),
            b_0, b_1, d_bpitch / sizeof(double),
            r_0, r_1, d_rpitch / sizeof(double));
    } else {
        broadcast_op_kernel<double><<<gridsize, blocksize>>>(d_A, d_B, d_R, h_op,
            a_0, a_1, d_apitch / sizeof(double),
            b_0, b_1, d_bpitch / sizeof(double),
            r_0, r_1, d_rpitch / sizeof(double));
    }
    hipDeviceSynchronize();
    cudaCheckError();

    std::cerr << "FINISHED !" << std::endl;

    // host result
    double *h_r = (double*)malloc(r_0 * d_rpitch);
    runtime_assert(h_r != nullptr, "Alloc error !");

    // copy back to host
    hipMemcpy(h_r, d_R, r_0 * d_rpitch, hipMemcpyDeviceToHost);
    cudaCheckError();

    // checking result
    std::cerr << cpuMean << std::endl;
    std::cerr << A << std::endl;
    double *h_Rcpu = A.getArray();
    runtime_assert(r_0 == A.getDim0() && r_1 == A.getDim1(), "Invalid shapes !");
    for (size_t i = 0; i < r_0; ++i)
    {
        for (size_t j = 0; j < r_1; ++j)
        {
	    std::cerr << h_r[i * (d_rpitch / sizeof(double)) + j] << " ";
            if (h_r[j + i * (d_rpitch / sizeof(double))] != h_Rcpu[j + i * r_1])
            {
                std::cerr << i << "," << j << " : Difference : "
                    << "GPU: " << h_r[j + i * (d_rpitch / sizeof(double))]
                    << std::endl
                    << "CPU: " << h_Rcpu[j + i * r_1]
                    << std::endl;
                return EXIT_FAILURE; // Free...
            }
        }
	std::cerr << std::endl;
    }

    std::cerr << "SUCCESS !" << std::endl;

    // free memory
    hipFree(d_A);
    cudaCheckError();
    hipFree(d_B);
    cudaCheckError();
    // in case not inplace:
    //hipFree(d_R);
    //cudaCheckError();
    free(h_r);
}
