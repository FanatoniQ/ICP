#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <iomanip>

// CPU
#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"
#include "libalg/broadcasting.hpp"
#include "error.hpp"

// GPU
#include "error.cuh"
#include "libgpualg/ope.cuh"


template <typename T>
__device__ func2_t<T> add2_op;

template <typename T>
__device__ func2_t<T> subtract2_op;

template <typename T>
__device__ func2_t<T> mult2_op;

template <typename T>
__device__ func2_t<T> divide2_op;



int main(int argc, char **argv)
{
    runtime_assert(argc == 2, "Usage: ./testgpuope file1");
    
    // retrieving functions (this part is not required if not on __host__ function)
    func2_t<double> h_add2_op, h_subtract2_op, h_mult2_op, h_divide2_op;
    hipMemcpyFromSymbol(&h_add2_op, HIP_SYMBOL(add2_op<double>), sizeof(func2_t<double>));
    hipMemcpyFromSymbol(&h_subtract2_op, HIP_SYMBOL(subtract2_op<double>), sizeof(func2_t<double>));
    hipMemcpyFromSymbol(&h_mult2_op, HIP_SYMBOL(mult2_op<double>), sizeof(func2_t<double>));
    hipMemcpyFromSymbol(&h_divide2_op, HIP_SYMBOL(divide2_op<double>), sizeof(func2_t<double>));

    // reading file, cpu operations
    std::string h{};
    size_t nblines, nbcols;
    double *h_A = readCSV(argv[1], h, nblines, nbcols);
    std::cerr << nblines << nbcols << std::endl;
    auto A = CPUMatrix(h_A, nblines, nbcols);
    std::cerr << A << std::endl;
    auto cpuMean = A.mean(0); //.transpose();
    auto R = A - cpuMean; // testing centered data

    // left operand
    double *d_A;
    size_t d_apitch;
    unsigned int a_0 = A.getDim0(), a_1 = A.getDim1(); //size_t width = nbcols, height = nblines;
    hipMallocPitch(&d_A, &d_apitch, a_1 * sizeof(double), a_0 * sizeof(double));
    cudaCheckError();
    hipMemcpy2D(d_A, d_apitch, A.getArray(), a_1 * sizeof(double), a_1 * sizeof(double), a_0, hipMemcpyHostToDevice);
    cudaCheckError();

    // right operand
    double *d_B;
    size_t d_bpitch;
    unsigned int b_0 = cpuMean.getDim0(), b_1 = cpuMean.getDim1();
    hipMallocPitch(&d_B, &d_bpitch, b_1 * sizeof(double), b_0 * sizeof(double));
    cudaCheckError();
    hipMemcpy2D(d_B, d_bpitch, cpuMean.getArray(), b_1 * sizeof(double), b_1 * sizeof(double), b_0, hipMemcpyHostToDevice);
    cudaCheckError();

    // result
    double *d_R = d_A; // in place operation
    size_t d_rpitch = d_apitch;
    size_t r_0, r_1;
    runtime_assert(get_broadcastable_size(a_0, a_1, b_0, b_1, &r_0, &r_1), "Invalid size for broadcasting !");
    runtime_assert(r_0 == a_0 && r_1 == a_1, "Invalid broadcasting for inplace operation !");

    // Launch the kernel
    dim3 blocksize(32,32); // 1024 threads per block TODO: change to test
    int nbblocksx = std::ceil((float)r_1 / blocksize.x);
    int nbblocksy = std::ceil((float)r_0 / blocksize.y);
    dim3 gridsize(nbblocksx, nbblocksy);
    runtime_assert(gridsize.x * gridsize.y * blocksize.x * blocksize.y >= r_0 * r_1, "Not enough threads !");
    //int threads = 4; // TODO: change this
    //int blocks = std::ceil((float)r_0 * r_1 / threads);
    //dim3 blocks(nbblocks, height);
    //broadcast_op_kernel<double><<<gridsize, blocksize>>>(d_A, d_B, d_R, h_subtract2_op,
    std::cerr << d_apitch << std::endl;
    broadcast_subtract_kernel<<<gridsize, blocksize>>>(d_A, d_B, d_R,
        a_0, a_1, d_apitch,
        b_0, b_1, d_bpitch,
        r_0, r_1, d_rpitch);
    hipDeviceSynchronize();
    cudaCheckError();

    // host result
    double *h_r = (double*)malloc(r_0 * d_rpitch);
    runtime_assert(h_r != nullptr, "Alloc error !");

    // copy back to host
    hipMemcpy(h_r, d_R, r_0 * d_rpitch, hipMemcpyDeviceToHost);
    cudaCheckError();

    // checking result
    std::cerr << cpuMean << std::endl;
    std::cerr << R << std::endl;
    double *h_Rcpu = R.getArray();
    runtime_assert(r_0 == R.getDim0() && r_1 == R.getDim1(), "Invalid shapes !");
    for (size_t i = 0; i < r_0; ++i)
    {
        for (size_t j = 0; j < r_1; ++j)
        {
	    std::cerr << h_r[i * d_rpitch + j] << " ";
	    /**
            if (h_r[j + i * d_rpitch] != h_Rcpu[j + i * r_1])
            {
                std::cerr << i << "," << j << " : Difference : "
                    << "GPU: " << h_r[j + i * d_rpitch]
                    << std::endl
                    << "CPU: " << h_Rcpu[j + i * r_1]
                    << std::endl;
                //return EXIT_FAILURE; // Free...
            }
	    **/
        }
	std::cerr << std::endl;
    }

    std::cerr << "SUCCESS !" << std::endl;

    // free memory
    hipFree(d_A);
    cudaCheckError();
    hipFree(d_B);
    cudaCheckError();
    // in case not inplace:
    //hipFree(d_R);
    //cudaCheckError();
    free(h_r);
}
