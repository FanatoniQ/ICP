#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <tuple>
#include <iostream>
#include <limits>
#include <float.h>
#include <assert.h>

// CPU
#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/CPUView.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"
#include "error.hpp"


// GPU
#include "libgpualg/mean.cuh"
#include "error.cuh"
//#include "gpu/icp.cuh"
#include "libgpuicp/dist.cuh"
#include "libgpuicp/corresp.cuh"

int main(int argc, char **argv)
{
    std::string f1Header{};
    size_t Qlines, Qcols, Plines, Pcols;
    double *Pt = readCSV(argv[1], f1Header, Plines, Pcols);
    CPUMatrix P = CPUMatrix(Pt, Plines, Pcols);
    double *Qt = readCSV(argv[2], f1Header, Qlines, Qcols);
    CPUMatrix Q = CPUMatrix(Qt, Qlines, Qcols);

    ICPCorresp *h_ref_dist = (ICPCorresp *)malloc(Plines * Qlines * sizeof(ICPCorresp));
    runtime_assert(h_ref_dist != nullptr, "Invalid ptr");
    for (size_t i = 0; i < Plines; ++i)
    {
         for (size_t j = 0; j < Qlines; ++j)
         {
             auto dist = P.getLine(i).euclidianDistance(Q.getLine(j));
	     assert(P.getLine(i).euclidianDistance(Q.getLine(j)) == Q.getLine(j).euclidianDistance(P.getLine(i)));
             h_ref_dist[i * Qlines + j] = {dist, (unsigned int)j};
         }
    }

    // device P matrix
    size_t p_pitch = Pcols * sizeof(double);
    double *d_P;
    //hipMallocPitch((void **)&d_P, &p_pitch, Pcols * sizeof(double), Plines);
    hipMalloc((void**)&d_P, Plines * p_pitch);
    cudaCheckError();
    hipMemcpy2D(d_P, p_pitch, Pt, Pcols * sizeof(double), Pcols * sizeof(double), Plines, hipMemcpyHostToDevice);
    cudaCheckError();

    // device Q matrix
    size_t q_pitch = Qcols * sizeof(double);
    double *d_Q;
    //hipMallocPitch((void **)&d_Q, &q_pitch, Qcols * sizeof(double), Qlines);
    hipMalloc((void**)&d_Q, Qlines * q_pitch);
    cudaCheckError();
    hipMemcpy2D(d_Q, q_pitch, Qt, Qcols * sizeof(double), Qcols * sizeof(double), Qlines, hipMemcpyHostToDevice);
    cudaCheckError();

    // device dist matrix
    size_t dist_pitch;
    ICPCorresp *d_dist;
    hipMallocPitch((void **)&d_dist, &dist_pitch, Qlines * sizeof(ICPCorresp), Plines);
    cudaCheckError();

    // call kernel
    get_distances(d_P, d_Q, &d_dist, Plines, Pcols, p_pitch, Qlines, Qcols, q_pitch, Plines, Qlines, &dist_pitch, true);
    std::cerr << "DONE" << std::endl;

    // copy back to host
    // host dist matrix
    ICPCorresp *h_dist = (ICPCorresp *)malloc(Plines * Qlines * sizeof(ICPCorresp));
    hipMemcpy2D(h_dist, Qlines * sizeof(ICPCorresp), d_dist, dist_pitch, Qlines * sizeof(ICPCorresp), Plines, hipMemcpyDeviceToHost);
    cudaCheckError();

    double ttlerror = 0;
    for (size_t i = 0; i < Plines; ++i)
    {
         for (size_t j = 0; j < Qlines; ++j)
         {
             //std::cerr << "dist: " << h_dist[i * Qlines + j].dist << " id: " <<  h_dist[i * Qlines + j].id << "\t";
	     if (h_dist[i * Qlines + j].id != h_ref_dist[i * Qlines + j].id)
	     {
		     std::cerr << "FATAL ID ERROR !" << std::endl;
		     return EXIT_FAILURE;
	     }
	     double err = std::fabs(h_dist[i * Qlines + j].dist - h_ref_dist[i * Qlines + j].dist);
	     ttlerror += err;
             std::cerr << err << "\t";
             //if (memcmp(&h_dist[i * Qlines + j], &h_ref_dist[i * Qlines + j], sizeof(ICPCorresp)) != 0)
             //{
                 //std::cerr << "h_ref_dist: " << h_ref_dist[i * Qlines + j].dist << " id: " << h_ref_dist[i * Qlines + j].id << std::endl;
		 //return EXIT_FAILURE;
             //}
         }
	 std::cerr << std::endl;
    }
    std::cerr << std::endl << "ttlerror: " << ttlerror << std::endl;
    std::cerr << "mean error: " << ttlerror / (Plines * Qlines) << std::endl;
    std::cerr << "SUCCESS" << std::endl;

    free(h_dist);
    free(h_ref_dist);
    hipFree(d_P);
    cudaCheckError();
    hipFree(d_Q);
    cudaCheckError();
    hipFree(d_dist);
    cudaCheckError();
}
