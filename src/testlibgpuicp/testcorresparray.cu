#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <tuple>
#include <iostream>
#include <limits>
#include <float.h>

// CPU
#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"
#include "error.hpp"


// GPU
#include "libgpualg/mean.cuh"
#include "error.cuh"
//#include "gpu/icp.cuh"
#include "libgpuicp/corresp.cuh"
#include "libgpuicp/crosscov.cuh"

int main(int argc, char **argv)
{
    std::string f1Header{};
    size_t Qlines, Qcols, Plines, Pcols;
    float* Pt = readCSV(argv[1], f1Header, Plines, Pcols);
    float* Qt = readCSV(argv[2], f1Header, Qlines, Qcols);
    float* d_P, * d_Q;

    unsigned int p_pitch = Pcols * sizeof(float);
    hipMalloc(&d_P, sizeof(float) * Plines * Pcols);
    unsigned int q_pitch = Qcols * sizeof(float);
    hipMalloc(&d_Q, sizeof(float) * Qlines * Qcols);

    hipMemcpy(d_P, Pt, sizeof(float) * Pcols * Plines, hipMemcpyHostToDevice);
    hipMemcpy(d_Q, Qt, sizeof(float) * Qcols * Qlines, hipMemcpyHostToDevice);

    unsigned int* d_array_correspondances;
    hipMalloc(&d_array_correspondances, sizeof(unsigned int) * Plines);

    get_array_correspondences(d_array_correspondances, d_P, d_Q, Plines, Pcols, Qlines, Qcols);

    unsigned int r_0 = Plines, r_1 = Pcols * Qcols;
    float* d_R = nullptr;
    size_t r_pitch;

    get_array_cross_covs_flattened(d_P, d_Q, &d_R, d_array_correspondances,
        Plines, Pcols, p_pitch,
        Qlines, Qcols, q_pitch,
        r_0, r_1, &r_pitch,
        Pcols, true);

    /**
    // Testing h_array_correspondances
    unsigned int* h_array_correspondances = (unsigned int *)malloc(Plines * sizeof(unsigned int));
    hipMemcpy(h_array_correspondances, d_array_correspondances, Plines * sizeof(unsigned int), hipMemcpyDeviceToHost);
    for (int i = 0; i < Plines; i++)
        std::cout << i << " ->" << h_array_correspondances[i] << std::endl;
    free(h_array_correspondances);
    **/

    hipFree(d_R);
    hipFree(d_P);
    hipFree(d_Q);
    hipFree(d_array_correspondances);
    free(Pt);
    free(Qt);
}
