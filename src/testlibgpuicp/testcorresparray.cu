#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <tuple>
#include <iostream>
#include <limits>
#include <float.h>

// CPU
#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"
#include "error.hpp"


// GPU
#include "libgpualg/mean.cuh"
#include "error.cuh"
//#include "gpu/icp.cuh"
#include "libgpuicp/corresp.cuh"

int main(int argc, char **argv)
{
    std::string f1Header{};
    size_t Qlines, Qcols, Plines, Pcols;
    //size_t Plines, Pcols;
    //___readCSV(f, f1Header);
    double* Pt = readCSV(argv[1], f1Header, Plines, Pcols);
    for (int i = 0; i < 30; i++)
        std::cout << Pt[i] << std::endl;
    double* Qt = readCSV(argv[2], f1Header, Qlines, Qcols);

    double* d_P, * d_Q;

    hipMalloc(&d_P, sizeof(double) * Plines * Pcols);
    hipMalloc(&d_Q, sizeof(double) * Qlines * Qcols);

    hipMemcpy(d_P, Pt, sizeof(double) * Pcols * Plines, hipMemcpyHostToDevice);
    hipMemcpy(d_Q, Qt, sizeof(double) * Qcols * Qlines, hipMemcpyHostToDevice);

    unsigned int* d_array_correspondances;
    hipMalloc(&d_array_correspondances, sizeof(unsigned int) * Plines);

    get_array_correspondences(d_array_correspondances, d_P, d_Q, Plines, Pcols, Qlines, Qcols);

    hipMemcpy(Qt, d_array_correspondances, sizeof(unsigned int) * Qcols, hipMemcpyDeviceToHost);

    for (int i = 0; i < 30; i++)
        std::cout << Qt[i] << std::endl;

    hipFree(d_P);
    hipFree(d_Q);
    hipFree(d_array_correspondances);
    free(Pt);
    free(Qt);
}
