#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <tuple>
#include <iostream>
#include <limits>
#include <float.h>

// CPU
#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"
#include "error.hpp"


// GPU
#include "libgpualg/mean.cuh"
#include "error.cuh"
//#include "gpu/icp.cuh"
#include "libgpuicp/corresp.cuh"
#include "libgpuicp/crosscov.cuh"

int main(int argc, char **argv)
{
    std::string f1Header{};
    size_t Qlines, Qcols, Plines, Pcols;
    double* Pt = readCSV(argv[1], f1Header, Plines, Pcols);
    double* Qt = readCSV(argv[2], f1Header, Qlines, Qcols);
    double* d_P, * d_Q;

    size_t p_pitch, q_pitch;
    if (strcmp(argv[3],"pitched") != 0) {
        std::cerr << "Non pitching alloc !" << std::endl;
        p_pitch = Pcols * sizeof(double);
        hipMalloc(&d_P, sizeof(double) * Plines * Pcols);
        q_pitch = Qcols * sizeof(double);
        hipMalloc(&d_Q, sizeof(double) * Qlines * Qcols);
    } else {
        std::cerr << "Pitching alloc !" << std::endl;
        hipMallocPitch(&d_P, &p_pitch, sizeof(double) * Pcols, Plines);
        hipMallocPitch(&d_Q, &q_pitch, sizeof(double) * Qcols, Qlines);
    }

    hipMemcpy(d_P, Pt, sizeof(double) * Pcols * Plines, hipMemcpyHostToDevice);
    hipMemcpy(d_Q, Qt, sizeof(double) * Qcols * Qlines, hipMemcpyHostToDevice);

    unsigned int* d_array_correspondances;
    hipMalloc(&d_array_correspondances, sizeof(unsigned int) * Plines);

    get_array_correspondences(d_array_correspondances, d_P, d_Q, Plines, Pcols, Qlines, Qcols, p_pitch, q_pitch);

    unsigned int r_0 = Plines, r_1 = Pcols * Qcols;
    double* d_R = nullptr;
    size_t r_pitch;

    get_array_cross_covs_flattened(d_P, d_Q, &d_R, d_array_correspondances,
        Plines, Pcols, p_pitch,
        Qlines, Qcols, q_pitch,
        r_0, r_1, &r_pitch,
        r_0, true);

    /**
    // Testing h_array_correspondances
    unsigned int* h_array_correspondances = (unsigned int *)malloc(Plines * sizeof(unsigned int));
    hipMemcpy(h_array_correspondances, d_array_correspondances, Plines * sizeof(unsigned int), hipMemcpyDeviceToHost);
    for (int i = 0; i < Plines; i++)
        std::cout << i << " ->" << h_array_correspondances[i] << std::endl;
    free(h_array_correspondances);
    **/

    hipFree(d_R);
    hipFree(d_P);
    hipFree(d_Q);
    hipFree(d_array_correspondances);
    free(Pt);
    free(Qt);
}
