#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <tuple>
#include <iostream>
#include <limits>
#include <float.h>
#include <assert.h>

// CPU
#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/CPUView.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"
#include "error.hpp"

// GPU
#include "libgpualg/ope.cuh"
#include "libgpualg/mean.cuh"
#include "error.cuh"
#include "libgpuicp/dist.cuh"
#include "libgpuicp/corresp.cuh"
#include "libgpuicp/crosscov.cuh"
#include "libgpuicp/batchcovs.cuh"

int main(int argc, char **argv)
{
    runtime_assert(argc == 3 || argc == 4, "./batchcov file1 file2 [batchsize]");
    std::string f1Header{};
    size_t Qlines, Qcols, Plines, Pcols;
    float *Pt = readCSV(argv[1], f1Header, Plines, Pcols);
    float *Qt = readCSV(argv[2], f1Header, Qlines, Qcols);

    size_t batchsize = 16;
    if (argc == 4)
        batchsize = std::stol(argv[3]);

    // device P matrix
    size_t p_pitch = Pcols * sizeof(float);
    float *d_P;
    //cudaMallodist_pitch((void **)&d_P, &p_pitch, Pcols * sizeof(float), Plines);
    hipMalloc((void**)&d_P, Plines * p_pitch);
    cudaCheckError();
    hipMemcpy2D(d_P, p_pitch, Pt, Pcols * sizeof(float), Pcols * sizeof(float), Plines, hipMemcpyHostToDevice);
    cudaCheckError();

    // device Q matrix
    size_t q_pitch = Qcols * sizeof(float);
    float *d_Q;
    //cudaMallodist_pitch((void **)&d_Q, &q_pitch, Qcols * sizeof(float), Qlines);
    hipMalloc((void**)&d_Q, Qlines * q_pitch);
    cudaCheckError();
    hipMemcpy2D(d_Q, q_pitch, Qt, Qcols * sizeof(float), Qcols * sizeof(float), Qlines, hipMemcpyHostToDevice);
    cudaCheckError();

    // device dist,id distance and corresp matrix
    size_t dist_0 = batchsize, dist_1 = Qlines;
    size_t dist_pitch;
    ICPCorresp *d_dist;
    hipMallocPitch((void **)&d_dist, &dist_pitch, dist_1 * sizeof(ICPCorresp), dist_0);
    cudaCheckError();
    //hipMemcpy2D(d_dist, dist_pitch, C, Qlines * sizeof(ICPCorresp), Qlines * sizeof(ICPCorresp), Plines, hipMemcpyHostToDevice);
    //cudaCheckError();

    // device cross-covs flattened
    size_t Rlines = batchsize, Rcols = Pcols * Qcols;
    size_t r_pitch = Rcols * sizeof(float);
    float *d_R;
    // or 2d...
    hipMalloc((void**)&d_R, Rlines * r_pitch);
    cudaCheckError();

    // device FINAL cross-cov (flattened)
    size_t covLines = Qcols, covCols = Pcols;
    size_t cov_pitch = covCols * covLines * sizeof(float);
    float *d_cov;
    hipMalloc((void**)&d_cov, 1 * cov_pitch);
    cudaCheckError();

    // CALL get_batch_cov
    get_batch_cov(d_P, Plines, Pcols, p_pitch,
        d_Q, Qlines, Qcols, q_pitch,
        d_dist, dist_0, dist_1, dist_pitch,
        d_R, Rlines, Rcols, r_pitch,
        d_cov, covLines, covCols, cov_pitch,
	batchsize
    );

    float *h_cov = (float *)malloc(covLines * covCols * sizeof(float));
    hipMemcpy(h_cov, d_cov, covLines * covCols * sizeof(float), hipMemcpyDeviceToHost);
    auto FULLGPUCOV = CPUMatrix(h_cov, covLines, covCols);
    std::cout << FULLGPUCOV << std::endl;

    hipFree(d_cov);
    cudaCheckError();
    hipFree(d_P);
    cudaCheckError();
    hipFree(d_Q);
    cudaCheckError();
    hipFree(d_R);
    cudaCheckError();
    hipFree(d_dist);
    cudaCheckError();
}
