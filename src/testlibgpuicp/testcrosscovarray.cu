#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <tuple>
#include <iostream>
#include <limits>
#include <float.h>
#include <assert.h>

// CPU
#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/CPUView.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"
#include "error.hpp"


// GPU
#include "libgpualg/mean.cuh"
#include "error.cuh"
//#include "gpu/icp.cuh"
#include "libgpuicp/corresp.cuh"
#include "libgpuicp/dist.cuh"
#include "libgpuicp/corresp.cuh"
#include "libgpuicp/crosscov.cuh"


int main(int argc, char **argv)
{
    std::string f1Header{};
    size_t Qlines, Qcols, Plines, Pcols;
    //size_t Plines, Pcols;
    //___readCSV(f, f1Header);
    float* Pt = readCSV(argv[1], f1Header, Plines, Pcols);
    float* Qt = readCSV(argv[2], f1Header, Qlines, Qcols);

    float* d_P, * d_Q, *d_cov;

    hipMalloc(&d_P, sizeof(float) * Plines * Pcols);
    hipMalloc(&d_Q, sizeof(float) * Qlines * Qcols);

    hipMemcpy(d_P, Pt, sizeof(float) * Pcols * Plines, hipMemcpyHostToDevice);
    hipMemcpy(d_Q, Qt, sizeof(float) * Qcols * Qlines, hipMemcpyHostToDevice);

    unsigned int* d_array_correspondances;
    hipMalloc(&d_array_correspondances, sizeof(unsigned int) * Plines);

    get_array_correspondences(d_array_correspondances, d_P, d_Q, Plines, Pcols, Qlines, Qcols);

    /*
    unsigned int* h_array_correspondances = (unsigned int*)malloc(Plines * sizeof(unsigned int));
    hipMemcpy(h_array_correspondances, d_array_correspondances, Plines * sizeof(unsigned int), hipMemcpyDeviceToHost);

    for (int i = 0; i < Plines; i++)
        std::cout << i << " ->" << h_array_correspondances[i] << std::endl;
    */

    hipMalloc(&d_cov, sizeof(float) * Qcols * Pcols);
    get_array_cross_cov(d_cov, d_array_correspondances, d_P, d_Q, Plines, Pcols, Qlines, Qcols);
    float* h_cov = (float*)malloc(Pcols * Qcols * sizeof(float));
    hipMemcpy(h_cov, d_cov, Pcols * Qcols * sizeof(unsigned int), hipMemcpyDeviceToHost);

    for (int i = 0; i < Pcols * Qcols; i++)
    {
        std::cout << h_cov[i] << " ";
        if (i % 3)
            std::cout << std::endl;
    }

    hipFree(d_P);
    hipFree(d_Q);
    hipFree(d_array_correspondances);
    free(h_cov);
    free(Pt);
    free(Qt);
}
