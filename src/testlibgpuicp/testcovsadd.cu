#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <tuple>
#include <iostream>
#include <limits>
#include <float.h>
#include <assert.h>

// CPU
#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/CPUView.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"
#include "error.hpp"


// GPU
#include "libgpualg/mean.cuh"
#include "error.cuh"
//#include "gpu/icp.cuh"
#include "libgpuicp/corresp.cuh"
#include "libgpuicp/dist.cuh"
#include "libgpuicp/corresp.cuh"
#include "libgpuicp/crosscov.cuh"

#define DISTS_LINES 128

__host__ double *get_cross_covs_cpu(CPUMatrix &P, size_t p_0, size_t p_1,
    CPUMatrix &Q, size_t q_0, size_t q_1,
    ICPCorresp *d_dist, size_t dist_0, size_t dist_1, size_t dist_pitch, size_t startindex)
{
    size_t ref_pitch = q_1 * p_1 * sizeof(double);
    double *h_ref = (double*)malloc(p_0 * ref_pitch);

    size_t h_dist_pitch = dist_1 * sizeof(ICPCorresp);
    ICPCorresp *h_dist = (ICPCorresp*)malloc(dist_0 * h_dist_pitch);

    hipMemcpy2D(h_dist, h_dist_pitch, d_dist, dist_pitch, dist_1 * sizeof(ICPCorresp), dist_0, hipMemcpyDeviceToHost);
    cudaCheckError();

    for (size_t i = 0; i < p_0; ++i)
    {
        size_t idp = i + startindex;
        size_t idq = h_dist[i * dist_1].id;
        std::cerr << "idq: " << idq << "idp: " << idp << std::endl;
        auto cov = Q.getLine(idq).transpose().dot(P.getLine(idp)); // since getLine returns line vector
        //std::cerr << ref_pitch << std::endl;
        //std::cerr << cov.getDim0() * cov.getDim1() * sizeof(double) << std::endl;
        assert(ref_pitch == (cov.getDim0() * cov.getDim1() * sizeof(double)));
        memcpy(h_ref + i * ref_pitch / sizeof(double), cov.getArray(), cov.getDim0() * cov.getDim1() * sizeof(double));
        for (size_t a = 0; a < q_1 * p_1; ++a)
            std::cerr << cov.getArray()[a] << "\t";
        std::cerr << std::endl;
    }

    free(h_dist);

    return h_ref;
}

int main(int argc, char **argv)
{
    runtime_assert(argc == 3, "./bin file1 file2");
    std::string f1Header{};
    size_t Qlines, Qcols, Plines, Pcols;
    double *Pt = readCSV(argv[1], f1Header, Plines, Pcols);
    CPUMatrix P = CPUMatrix(Pt, Plines, Pcols);
    double *Qt = readCSV(argv[2], f1Header, Qlines, Qcols);
    CPUMatrix Q = CPUMatrix(Qt, Qlines, Qcols);

    // device P matrix
    size_t p_pitch = Pcols * sizeof(double);
    double *d_P;
    //cudaMallodist_pitch((void **)&d_P, &p_pitch, Pcols * sizeof(double), Plines);
    hipMalloc((void**)&d_P, Plines * p_pitch);
    cudaCheckError();
    hipMemcpy2D(d_P, p_pitch, Pt, Pcols * sizeof(double), Pcols * sizeof(double), Plines, hipMemcpyHostToDevice);
    cudaCheckError();

    // device Q matrix
    size_t q_pitch = Qcols * sizeof(double);
    double *d_Q;
    //cudaMallodist_pitch((void **)&d_Q, &q_pitch, Qcols * sizeof(double), Qlines);
    hipMalloc((void**)&d_Q, Qlines * q_pitch);
    cudaCheckError();
    hipMemcpy2D(d_Q, q_pitch, Qt, Qcols * sizeof(double), Qcols * sizeof(double), Qlines, hipMemcpyHostToDevice);
    cudaCheckError();

    //size_t nbiters =  std::ceil((float)Plines / DISTS_LINES);
    size_t Pstartindex = 0;
    size_t nblines;

    // device dist,id distance and corresp matrix
    size_t dist_0 = DISTS_LINES, dist_1 = Qlines;
    size_t dist_pitch;
    ICPCorresp *d_dist;
    hipMallocPitch((void **)&d_dist, &dist_pitch, dist_1 * sizeof(ICPCorresp), dist_0);
    cudaCheckError();
    //hipMemcpy2D(d_dist, dist_pitch, C, Qlines * sizeof(ICPCorresp), Qlines * sizeof(ICPCorresp), Plines, hipMemcpyHostToDevice);
    //cudaCheckError();

    // device cross-covs flattened
    size_t Rlines = DISTS_LINES, Rcols = Pcols * Qcols;
    size_t r_pitch = Rcols * sizeof(double);
    double *d_R;
    // or 2d...
    hipMalloc((void**)&d_R, Rlines * r_pitch);
    cudaCheckError();

    // device FINAL cross-cov (flattened)
    size_t covLines = Qcols, covCols = Pcols;
    size_t cov_pitch = covCols * covLines * sizeof(double);
    double *d_cov;
    hipMalloc((void**)&d_cov, 1 * cov_pitch);
    hipMemset(d_cov, 0, cov_pitch);
    cudaCheckError();

    double ttlerror = 0;
    auto COV = CPUMatrix(Qcols, Pcols);
    auto RefCOV = CPUMatrix(Qcols, Pcols);

    while (Pstartindex < Plines)
    {
        nblines = MIN(Plines - Pstartindex, DISTS_LINES);

        // DISTS
        get_distances(d_P + Pstartindex * p_pitch / sizeof(double), d_Q, &d_dist, nblines, Pcols, p_pitch, Qlines, Qcols, q_pitch, nblines, dist_1, &dist_pitch, true);
        std::cerr << "DISTS DONE" << std::endl;

        // CORRESPS
        get_correspondences(d_dist, dist_pitch, nblines, dist_1, true);
        std::cerr << "CORRESPS DONE" << std::endl;

        /** Testing corresps: **/
        ICPCorresp *h_corresp = (ICPCorresp *)malloc(nblines * 1 * sizeof(ICPCorresp));
        hipMemcpy2D(h_corresp, 1 * sizeof(ICPCorresp), d_dist, dist_pitch, 1 * sizeof(ICPCorresp), nblines, hipMemcpyDeviceToHost);
        cudaCheckError();
	//if (Pstartindex >= Plines - 100)
	if (Pstartindex < 100)
{	for (size_t i = 0; i < nblines; ++i)
	{
	//if (i + Pstartindex >= Plines - 100)
            std::cout << i + Pstartindex << " -> " << h_corresp[i].id << " , " << h_corresp[i].dist << std::endl;
	}
}
	free(h_corresp);


        // CROSS-COVS
        get_cross_cov(d_P + Pstartindex * p_pitch / sizeof(double), d_Q, &d_R, d_dist,
            nblines, Pcols, p_pitch,
            Qlines, Qcols, q_pitch,
            nblines, Rcols, &r_pitch,
            nblines, dist_1, dist_pitch, true);
        std::cerr << "CROSS-COVS DONE" << std::endl;

        /** Testing cross-covs: **/
        double *h_ref_cross_covs = get_cross_covs_cpu(P, nblines, Pcols, Q, Qlines, Qcols, d_dist, nblines, dist_1, dist_pitch, Pstartindex);
        double *h_r = (double*)malloc(nblines * Rcols * sizeof(double));
        hipMemcpy2D(h_r, Rcols * sizeof(double), d_R, r_pitch, Rcols * sizeof(double), nblines, hipMemcpyDeviceToHost);
        cudaCheckError();
        //auto BatchRefCOV = CPUMatrix(h_ref_cross_covs, Qcols, Pcols);
        auto BatchRefCOV = CPUMatrix(Qcols, Pcols);
        //assert(memcmp(h_ref_cross_covs, h_r, Rlines * Rcols * sizeof(double)) == 0);
	ttlerror = 0;
        for (size_t i = 0; i < nblines; i++)
        {
            for (size_t j = 0; j < Rcols; ++j)
            {
                double error = std::fabs(h_r[i * (r_pitch / sizeof(double)) + j] - h_ref_cross_covs[i * (r_pitch / sizeof(double)) + j]); // Weird not having to divide by sizeof double...
                std::cerr << h_r[i * (r_pitch / sizeof(double)) + j] << " \t " <<  h_ref_cross_covs[i * (r_pitch / sizeof(double)) + j] << std::endl;
                ttlerror += error;
            }
	    auto LineRefCOV = CPUMatrix(h_ref_cross_covs + i * (r_pitch / sizeof(double)), Qcols, Pcols);
	    BatchRefCOV += LineRefCOV;
	    LineRefCOV.setArray(nullptr,1,1);
        }
        std::cerr << "Error (batch cross-covs): " << ttlerror << std::endl;
        std::cerr << "Mean Error (batch cross-covs): " << ttlerror / nblines * Rcols << std::endl;
	//assert(Qcols * Pcols == nblines * Rcols); // should break
        RefCOV += BatchRefCOV;
        free(h_r);

        // COVS SUM
	//hipMemset(d_R, 0, r_pitch * Rlines);
        //reduce_0(MatrixReduceOP::SUM, d_dist, double **d_sum, Pcols * Qcols, Plines, dist_pitch, size_t *reducepitch, int threads);
        reduce_0(MatrixReduceOP::SUM, d_R, &d_R, Rcols, nblines, r_pitch, &r_pitch, nblines);

        // COV += COVS SUM
        assert(covCols == 3 && covLines == 3);
        matrix_op<double>(gridsize(1, 1), dim3(covCols * covLines,1), d_cov, d_R, d_cov, MatrixOP::ADD,
             1, covCols * covLines, cov_pitch,
             1, Rcols, r_pitch,
             1, covCols * covLines, cov_pitch);

        /** testing covs-sum **/
        /**for (size_t i = 0; i < Rlines; i++)
        {
            auto c = CPUMatrix(h_ref_cross_covs + i * (r_pitch / sizeof(double)), Qcols, Pcols);
        RefCOV += c;
        c.setArray(nullptr,1,1); // avoid freeing
        }**/
        // TODO: do this on GPU
        double *h_cov = (double *)malloc(Rcols * sizeof(double));
        hipMemcpy(h_cov, d_R, Rcols * sizeof(double), hipMemcpyDeviceToHost);
        auto BatchCOV = CPUMatrix(h_cov, Qcols, Pcols);
        COV += BatchCOV;

	ttlerror = 0;
        for (size_t i = 0; i < Qcols; ++i)
        {
            for (size_t j = 0; j < Pcols; ++j)
            {
                double error = std::fabs(BatchCOV(i,j) - BatchRefCOV(i,j));
				//h_cov[i * Pcols + j] - h_ref_cross_covs[i * (r_pitch / sizeof(double)) + j]); // Weird not having to divide by sizeof double...
                //std::cerr << h_r[i * (r_pitch / sizeof(double)) + j] << " \t " <<  h_ref_cross_covs[i * (r_pitch / sizeof(double)) + j] << std::endl;
                ttlerror += error;
            }
        }
	std::cout << "Error (batch-reduced cross-cov): " << ttlerror << std::endl;
        std::cout << "Mean Error (batch-reduced cross-cov): " << ttlerror / (Pcols * Qcols) << std::endl;

        std::cerr << "BatchRefCOV:" << std::endl;
        std::cerr << BatchRefCOV << std::endl;

        std::cerr << "BatchCOV:" << std::endl;
        std::cerr << BatchCOV << std::endl;

	//assert(BatchRefCOV.getArray() == BatchCOV.getArray());
	std::cout << "index: " << Pstartindex << "/" << Plines << std::endl;

        Pstartindex += nblines;
    }
    //auto COV = CPUMatrix(h_cov, Qcols, Pcols);
    ttlerror = 0;
    for (size_t i = 0; i < Qcols; ++i)
    {
        for (size_t j = 0; j < Pcols; ++j)
        {
            double error = std::fabs(RefCOV(i,j) - COV(i,j));
            ttlerror += error;
        }
    }
    std::cout << "Error (FINAL CPU summed cross-cov): " << ttlerror << std::endl;
    std::cout << "Mean Error (FINAL CPU summed cross-cov): " << ttlerror / (Pcols * Qcols) << std::endl;

    std::cout << "CPURefCOV:" << std::endl;
    std::cout << RefCOV << std::endl;

    std::cout << "CPUsummed GPUCOVs:" << std::endl;
    std::cout << COV << std::endl;

    double *h_cov = (double *)malloc(covLines * covCols * sizeof(double));
    hipMemcpy(h_cov, d_cov, covLines * covCols * sizeof(double), hipMemcpyDeviceToHost);
    auto FULLGPUCOV = CPUMatrix(h_cov, covLines, covCols);

    std::cout << "FULL GPUCOV:" << std::endl;
    std::cout << FULLGPUCOV << std::endl;
    
    hipFree(d_cov);
    cudaCheckError();
    hipFree(d_P);
    cudaCheckError();
    hipFree(d_Q);
    cudaCheckError();
    hipFree(d_R);
    cudaCheckError();
    hipFree(d_dist);
    cudaCheckError();
}
