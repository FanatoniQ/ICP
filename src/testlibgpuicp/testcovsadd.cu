#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <tuple>
#include <iostream>
#include <limits>
#include <float.h>
#include <assert.h>

// CPU
#include "libCSV/csv.hpp"
#include "libalg/CPUMatrix.hpp"
#include "libalg/CPUView.hpp"
#include "libalg/alg.hpp"
#include "libalg/print.hpp"
#include "error.hpp"


// GPU
#include "libgpualg/ope.cuh"
#include "libgpualg/mean.cuh"
#include "error.cuh"
//#include "gpu/icp.cuh"
#include "libgpuicp/corresp.cuh"
#include "libgpuicp/dist.cuh"
#include "libgpuicp/corresp.cuh"
#include "libgpuicp/crosscov.cuh"

#define DISTS_LINES 128

__host__ float *get_cross_covs_cpu(CPUMatrix &P, size_t p_0, size_t p_1,
    CPUMatrix &Q, size_t q_0, size_t q_1,
    ICPCorresp *d_dist, size_t dist_0, size_t dist_1, size_t dist_pitch, size_t startindex)
{
    size_t ref_pitch = q_1 * p_1 * sizeof(float);
    float *h_ref = (float*)malloc(p_0 * ref_pitch);

    size_t h_dist_pitch = dist_1 * sizeof(ICPCorresp);
    ICPCorresp *h_dist = (ICPCorresp*)malloc(dist_0 * h_dist_pitch);

    hipMemcpy2D(h_dist, h_dist_pitch, d_dist, dist_pitch, dist_1 * sizeof(ICPCorresp), dist_0, hipMemcpyDeviceToHost);
    cudaCheckError();

    for (size_t i = 0; i < p_0; ++i)
    {
        size_t idp = i + startindex;
        size_t idq = h_dist[i * dist_1].id;
        std::cerr << "idq: " << idq << "idp: " << idp << std::endl;
        auto cov = Q.getLine(idq).transpose().dot(P.getLine(idp)); // since getLine returns line vector
        //std::cerr << ref_pitch << std::endl;
        //std::cerr << cov.getDim0() * cov.getDim1() * sizeof(float) << std::endl;
        assert(ref_pitch == (cov.getDim0() * cov.getDim1() * sizeof(float)));
        memcpy(h_ref + i * ref_pitch / sizeof(float), cov.getArray(), cov.getDim0() * cov.getDim1() * sizeof(float));
        for (size_t a = 0; a < q_1 * p_1; ++a)
            std::cerr << cov.getArray()[a] << "\t";
        std::cerr << std::endl;
    }

    free(h_dist);

    return h_ref;
}

int main(int argc, char **argv)
{
    runtime_assert(argc == 3, "./bin file1 file2");
    std::string f1Header{};
    size_t Qlines, Qcols, Plines, Pcols;
    float *Pt = readCSV(argv[1], f1Header, Plines, Pcols);
    CPUMatrix P = CPUMatrix(Pt, Plines, Pcols);
    float *Qt = readCSV(argv[2], f1Header, Qlines, Qcols);
    CPUMatrix Q = CPUMatrix(Qt, Qlines, Qcols);

    // device P matrix
    size_t p_pitch = Pcols * sizeof(float);
    float *d_P;
    //cudaMallodist_pitch((void **)&d_P, &p_pitch, Pcols * sizeof(float), Plines);
    hipMalloc((void**)&d_P, Plines * p_pitch);
    cudaCheckError();
    hipMemcpy2D(d_P, p_pitch, Pt, Pcols * sizeof(float), Pcols * sizeof(float), Plines, hipMemcpyHostToDevice);
    cudaCheckError();

    // device Q matrix
    size_t q_pitch = Qcols * sizeof(float);
    float *d_Q;
    //cudaMallodist_pitch((void **)&d_Q, &q_pitch, Qcols * sizeof(float), Qlines);
    hipMalloc((void**)&d_Q, Qlines * q_pitch);
    cudaCheckError();
    hipMemcpy2D(d_Q, q_pitch, Qt, Qcols * sizeof(float), Qcols * sizeof(float), Qlines, hipMemcpyHostToDevice);
    cudaCheckError();

    //size_t nbiters =  std::ceil((float)Plines / DISTS_LINES);
    size_t Pstartindex = 0;
    size_t nblines;

    // device dist,id distance and corresp matrix
    size_t dist_0 = DISTS_LINES, dist_1 = Qlines;
    size_t dist_pitch;
    ICPCorresp *d_dist;
    hipMallocPitch((void **)&d_dist, &dist_pitch, dist_1 * sizeof(ICPCorresp), dist_0);
    cudaCheckError();
    //hipMemcpy2D(d_dist, dist_pitch, C, Qlines * sizeof(ICPCorresp), Qlines * sizeof(ICPCorresp), Plines, hipMemcpyHostToDevice);
    //cudaCheckError();

    // device cross-covs flattened
    size_t Rlines = DISTS_LINES, Rcols = Pcols * Qcols;
    size_t r_pitch = Rcols * sizeof(float);
    float *d_R;
    // or 2d...
    hipMalloc((void**)&d_R, Rlines * r_pitch);
    cudaCheckError();

    // device FINAL cross-cov (flattened)
    size_t covLines = Qcols, covCols = Pcols;
    size_t cov_pitch = covCols * covLines * sizeof(float);
    float *d_cov;
    hipMalloc((void**)&d_cov, 1 * cov_pitch);
    hipMemset(d_cov, 0, cov_pitch);
    cudaCheckError();

    float ttlerror = 0;
    auto COV = CPUMatrix(Qcols, Pcols);
    auto RefCOV = CPUMatrix(Qcols, Pcols);

    while (Pstartindex < Plines)
    {
        nblines = MIN(Plines - Pstartindex, DISTS_LINES);

        // DISTS
        get_distances(d_P + Pstartindex * p_pitch / sizeof(float), d_Q, &d_dist, nblines, Pcols, p_pitch, Qlines, Qcols, q_pitch, nblines, dist_1, &dist_pitch, true);
        std::cerr << "DISTS DONE" << std::endl;

        // CORRESPS
        get_correspondences(d_dist, dist_pitch, nblines, dist_1, true);
        std::cerr << "CORRESPS DONE" << std::endl;

        /** Testing corresps: **/
        ICPCorresp *h_corresp = (ICPCorresp *)malloc(nblines * 1 * sizeof(ICPCorresp));
        hipMemcpy2D(h_corresp, 1 * sizeof(ICPCorresp), d_dist, dist_pitch, 1 * sizeof(ICPCorresp), nblines, hipMemcpyDeviceToHost);
        cudaCheckError();
	//if (Pstartindex >= Plines - 100)
	if (Pstartindex < 100)
{	for (size_t i = 0; i < nblines; ++i)
	{
	//if (i + Pstartindex >= Plines - 100)
            std::cout << i + Pstartindex << " -> " << h_corresp[i].id << " , " << h_corresp[i].dist << std::endl;
	}
}
	free(h_corresp);


        // CROSS-COVS
        get_cross_cov(d_P + Pstartindex * p_pitch / sizeof(float), d_Q, &d_R, d_dist,
            nblines, Pcols, p_pitch,
            Qlines, Qcols, q_pitch,
            nblines, Rcols, &r_pitch,
            nblines, dist_1, dist_pitch, true);
        std::cerr << "CROSS-COVS DONE" << std::endl;

        /** Testing cross-covs: **/
        float *h_ref_cross_covs = get_cross_covs_cpu(P, nblines, Pcols, Q, Qlines, Qcols, d_dist, nblines, dist_1, dist_pitch, Pstartindex);
        float *h_r = (float*)malloc(nblines * Rcols * sizeof(float));
        hipMemcpy2D(h_r, Rcols * sizeof(float), d_R, r_pitch, Rcols * sizeof(float), nblines, hipMemcpyDeviceToHost);
        cudaCheckError();
        //auto BatchRefCOV = CPUMatrix(h_ref_cross_covs, Qcols, Pcols);
        auto BatchRefCOV = CPUMatrix(Qcols, Pcols);
        //assert(memcmp(h_ref_cross_covs, h_r, Rlines * Rcols * sizeof(float)) == 0);
	ttlerror = 0;
        for (size_t i = 0; i < nblines; i++)
        {
            for (size_t j = 0; j < Rcols; ++j)
            {
                float error = std::fabs(h_r[i * (r_pitch / sizeof(float)) + j] - h_ref_cross_covs[i * (r_pitch / sizeof(float)) + j]); // Weird not having to divide by sizeof float...
                std::cerr << h_r[i * (r_pitch / sizeof(float)) + j] << " \t " <<  h_ref_cross_covs[i * (r_pitch / sizeof(float)) + j] << std::endl;
                ttlerror += error;
            }
	    auto LineRefCOV = CPUMatrix(h_ref_cross_covs + i * (r_pitch / sizeof(float)), Qcols, Pcols);
	    BatchRefCOV += LineRefCOV;
	    LineRefCOV.setArray(nullptr,1,1);
        }
        std::cerr << "Error (batch cross-covs): " << ttlerror << std::endl;
        std::cerr << "Mean Error (batch cross-covs): " << ttlerror / nblines * Rcols << std::endl;
	//assert(Qcols * Pcols == nblines * Rcols); // should break
        RefCOV += BatchRefCOV;
        free(h_r);

        // COVS SUM
	//hipMemset(d_R, 0, r_pitch * Rlines);
        //reduce_0(MatrixReduceOP::SUM, d_dist, float **d_sum, Pcols * Qcols, Plines, dist_pitch, size_t *reducepitch, int threads);
        reduce_0(MatrixReduceOP::SUM, d_R, &d_R, Rcols, nblines, r_pitch, &r_pitch, nblines);

        // COV += COVS SUM
        assert(covCols == 3 && covLines == 3);
        matrix_op<float>(dim3(1, 1), dim3(covCols * covLines,1), d_cov, d_R, d_cov, MatrixOP::ADD,
             1, covCols * covLines, cov_pitch,
             1, Rcols, r_pitch,
             1, covCols * covLines, cov_pitch);

        /** testing covs-sum **/
        /**for (size_t i = 0; i < Rlines; i++)
        {
            auto c = CPUMatrix(h_ref_cross_covs + i * (r_pitch / sizeof(float)), Qcols, Pcols);
        RefCOV += c;
        c.setArray(nullptr,1,1); // avoid freeing
        }**/
        // TODO: do this on GPU
        float *h_cov = (float *)malloc(Rcols * sizeof(float));
        hipMemcpy(h_cov, d_R, Rcols * sizeof(float), hipMemcpyDeviceToHost);
        auto BatchCOV = CPUMatrix(h_cov, Qcols, Pcols);
        COV += BatchCOV;

	ttlerror = 0;
        for (size_t i = 0; i < Qcols; ++i)
        {
            for (size_t j = 0; j < Pcols; ++j)
            {
                float error = std::fabs(BatchCOV(i,j) - BatchRefCOV(i,j));
				//h_cov[i * Pcols + j] - h_ref_cross_covs[i * (r_pitch / sizeof(float)) + j]); // Weird not having to divide by sizeof float...
                //std::cerr << h_r[i * (r_pitch / sizeof(float)) + j] << " \t " <<  h_ref_cross_covs[i * (r_pitch / sizeof(float)) + j] << std::endl;
                ttlerror += error;
            }
        }
	std::cout << "Error (batch-reduced cross-cov): " << ttlerror << std::endl;
        std::cout << "Mean Error (batch-reduced cross-cov): " << ttlerror / (Pcols * Qcols) << std::endl;

        std::cerr << "BatchRefCOV:" << std::endl;
        std::cerr << BatchRefCOV << std::endl;

        std::cerr << "BatchCOV:" << std::endl;
        std::cerr << BatchCOV << std::endl;

	//assert(BatchRefCOV.getArray() == BatchCOV.getArray());
	std::cout << "index: " << Pstartindex << "/" << Plines << std::endl;

        Pstartindex += nblines;
    }
    //auto COV = CPUMatrix(h_cov, Qcols, Pcols);
    ttlerror = 0;
    for (size_t i = 0; i < Qcols; ++i)
    {
        for (size_t j = 0; j < Pcols; ++j)
        {
            float error = std::fabs(RefCOV(i,j) - COV(i,j));
            ttlerror += error;
        }
    }
    std::cout << "Error (FINAL CPU summed cross-cov): " << ttlerror << std::endl;
    std::cout << "Mean Error (FINAL CPU summed cross-cov): " << ttlerror / (Pcols * Qcols) << std::endl;

    std::cout << "CPURefCOV:" << std::endl;
    std::cout << RefCOV << std::endl;

    std::cout << "CPUsummed GPUCOVs:" << std::endl;
    std::cout << COV << std::endl;

    float *h_cov = (float *)malloc(covLines * covCols * sizeof(float));
    hipMemcpy(h_cov, d_cov, covLines * covCols * sizeof(float), hipMemcpyDeviceToHost);
    auto FULLGPUCOV = CPUMatrix(h_cov, covLines, covCols);
    
    ttlerror = 0;
    for (size_t i = 0; i < Qcols; ++i)
    {
        for (size_t j = 0; j < Pcols; ++j)
        {
            float error = std::fabs(RefCOV(i,j) - FULLGPUCOV(i,j));
            ttlerror += error;
        }
    }
    std::cout << "Error (FINAL FULLGPU cross-cov): " << ttlerror << std::endl;
    std::cout << "Mean Error (FINAL FULLGPU cross-cov): " << ttlerror / (Pcols * Qcols) << std::endl;

    std::cout << "FULL GPUCOV:" << std::endl;
    std::cout << FULLGPUCOV << std::endl;
    
    hipFree(d_cov);
    cudaCheckError();
    hipFree(d_P);
    cudaCheckError();
    hipFree(d_Q);
    cudaCheckError();
    hipFree(d_R);
    cudaCheckError();
    hipFree(d_dist);
    cudaCheckError();
}
